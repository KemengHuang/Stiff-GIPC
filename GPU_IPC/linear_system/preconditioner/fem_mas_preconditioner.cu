#include "hip/hip_runtime.h"
#include <linear_system/preconditioner/fem_mas_preconditioner.h>
#include <linear_system/subsystem/fem_linear_subsystem.h>
#include <gipc/utils/timer.h>
namespace gipc
{
MAS_Preconditioner::MAS_Preconditioner(FEMLinearSubsystem& subsystem,
                                       BHessian&           mBH,
                                       MASPreconditioner&  mMAS,
                                       double*             mMasses,
                                       uint32_t*           mCpNum)
    : Base(subsystem)
    , BH(mBH)
    , MAS_Prec(mMAS)
    , masses(mMasses)
    , cpNum(mCpNum)
{
}

void MAS_Preconditioner::assemble()
{
    double collision_num = *cpNum;
    gipc::Timer timer{"precomputing mas Preconditioner"};
    //MAS_Prec.setPreconditioner(BH, masses, collision_num);
    MAS_Prec.setPreconditioner_bcoo(system_bcoo_matrix(),
                                    calculate_subsystem_bcoo_indices(), get_offset(), collision_num);
}

void MAS_Preconditioner::apply(muda::CDenseVectorView<Float> r,
                              muda::DenseVectorView<Float>  z)
{

    MAS_Prec.preconditioning((double3*)r.data(), (double3*)z.data());

    //using namespace muda;

    //auto abd_body_count = m_sim_data.abd_fem_count_info().abd_body_num;
    //auto abd_inv_diag   = m_abd.abd_system_diag_preconditioner.view();

    //ParallelFor()
    //    .kernel_name(__FUNCTION__)
    //    .apply(abd_body_count,
    //           [r = r.viewer().name("r"),
    //            z = z.viewer().name("z"),
    //            inv = abd_inv_diag.viewer().name("inv")] __device__(int i) mutable
    //           {
    //               z.segment<12>(i * 12).as_eigen() =
    //                   inv(i) * r.segment<12>(i * 12).as_eigen();
    //           });
}
}  // namespace gipc
