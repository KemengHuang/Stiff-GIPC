#include "hip/hip_runtime.h"
//
// device_fem_data.cu
// GIPC
//
// created by Kemeng Huang on 2022/12/01
// Copyright (c) 2024 Kemeng Huang. All rights reserved.
//

#include "device_fem_data.cuh"
#include "cuda_tools.h"


void device_TetraData::Malloc_DEVICE_MEM(const int& vertex_num,
                                         const int& tetradedra_num,
                                         const int& triangle_num,
                                         const int& softNum,
                                         const int& tri_edgeNum,
                                         const int& bodyNum)
{
    int maxNumbers = vertex_num > tetradedra_num ? vertex_num : tetradedra_num;
    CUDA_SAFE_CALL(hipMalloc((void**)&vertexes, vertex_num * sizeof(double3)));
    CUDA_SAFE_CALL(hipMalloc((void**)&o_vertexes, vertex_num * sizeof(double3)));
    CUDA_SAFE_CALL(hipMalloc((void**)&velocities, vertex_num * sizeof(double3)));
    CUDA_SAFE_CALL(hipMalloc((void**)&rest_vertexes, vertex_num * sizeof(double3)));
    CUDA_SAFE_CALL(hipMalloc((void**)&temp_double3Mem, vertex_num * sizeof(double3)));
    CUDA_SAFE_CALL(hipMalloc((void**)&xTilta, vertex_num * sizeof(double3)));
    CUDA_SAFE_CALL(hipMalloc((void**)&fb, vertex_num * sizeof(double3)));
    CUDA_SAFE_CALL(hipMalloc((void**)&totalForce, vertex_num * sizeof(double3)));
    CUDA_SAFE_CALL(hipMalloc((void**)&shape_grads, vertex_num * sizeof(double3)));

    CUDA_SAFE_CALL(hipMalloc((void**)&tetrahedras, tetradedra_num * sizeof(uint4)));
    CUDA_SAFE_CALL(hipMalloc((void**)&tempTetrahedras, tetradedra_num * sizeof(uint4)));


    CUDA_SAFE_CALL(hipMalloc((void**)&tri_edges, tri_edgeNum * sizeof(uint2)));
    CUDA_SAFE_CALL(hipMalloc((void**)&tri_edge_adj_vertex, tri_edgeNum * sizeof(uint2)));


    CUDA_SAFE_CALL(hipMalloc((void**)&volum, tetradedra_num * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc((void**)&masses, vertex_num * sizeof(double)));

    CUDA_SAFE_CALL(hipMalloc((void**)&lengthRate, tetradedra_num * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc((void**)&volumeRate, tetradedra_num * sizeof(double)));

    CUDA_SAFE_CALL(hipMalloc((void**)&tempDouble, maxNumbers * sizeof(double)));
    //CUDA_SAFE_CALL(hipMalloc((void**)&tempM, vertex_num * sizeof(double)));

    CUDA_SAFE_CALL(hipMalloc((void**)&MChash, maxNumbers * sizeof(uint64_t)));
    CUDA_SAFE_CALL(hipMalloc((void**)&sortIndex, maxNumbers * sizeof(uint32_t)));
    CUDA_SAFE_CALL(hipMalloc((void**)&BoundaryType, vertex_num * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&tempBoundaryType, vertex_num * sizeof(int)));

    CUDA_SAFE_CALL(hipMemset(BoundaryType, 0, vertex_num * sizeof(int)));

    //CUDA_SAFE_CALL(hipMalloc((void**)&sortVertIndex, vertex_num * sizeof(uint32_t)));
    CUDA_SAFE_CALL(hipMalloc((void**)&sortMapVertIndex, vertex_num * sizeof(uint32_t)));

    CUDA_SAFE_CALL(hipMalloc((void**)&DmInverses,
                              tetradedra_num * sizeof(__GEIGEN__::Matrix3x3d)));
    CUDA_SAFE_CALL(hipMalloc((void**)&Constraints,
                              vertex_num * sizeof(__GEIGEN__::Matrix3x3d)));

    CUDA_SAFE_CALL(hipMalloc((void**)&tempMat3x3,
                              maxNumbers * sizeof(__GEIGEN__::Matrix3x3d)));
    //CUDA_SAFE_CALL(hipMalloc((void**)&tempConstraints, vertex_num * sizeof(__GEIGEN__::Matrix3x3d)));


    CUDA_SAFE_CALL(hipMalloc((void**)&targetIndex, softNum * sizeof(uint32_t)));
    CUDA_SAFE_CALL(hipMalloc((void**)&targetVert, softNum * sizeof(double3)));
    CUDA_SAFE_CALL(hipMalloc((void**)&triDmInverses,
                              triangle_num * sizeof(__GEIGEN__::Matrix2x2d)));


    CUDA_SAFE_CALL(hipMalloc((void**)&svd3x2F,
                              triangle_num * sizeof(Eigen::Matrix<double, 3, 2>)));
    CUDA_SAFE_CALL(hipMalloc((void**)&svd3x2U, triangle_num * sizeof(Eigen::Matrix3d)));
    CUDA_SAFE_CALL(hipMalloc((void**)&svd3x2V, triangle_num * sizeof(Eigen::Matrix2d)));
    CUDA_SAFE_CALL(hipMalloc((void**)&svd3x2S, triangle_num * sizeof(Eigen::Vector2d)));


    CUDA_SAFE_CALL(hipMalloc((void**)&area, triangle_num * sizeof(double)));
    CUDA_SAFE_CALL(hipMalloc((void**)&triangles, triangle_num * sizeof(uint4)));


    CUDA_SAFE_CALL(hipMalloc((void**)&body_id_to_boundary_type, bodyNum * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&point_id_to_body_id, vertex_num * sizeof(int)));
    CUDA_SAFE_CALL(hipMalloc((void**)&tet_id_to_body_id, tetradedra_num * sizeof(int)));
}

device_TetraData::~device_TetraData()
{
    FREE_DEVICE_MEM();
}

void device_TetraData::FREE_DEVICE_MEM()
{
    CUDA_SAFE_CALL(hipFree(sortIndex));
    CUDA_SAFE_CALL(hipFree(sortMapVertIndex));
    CUDA_SAFE_CALL(hipFree(vertexes));
    CUDA_SAFE_CALL(hipFree(o_vertexes));
    CUDA_SAFE_CALL(hipFree(temp_double3Mem));
    CUDA_SAFE_CALL(hipFree(velocities));
    CUDA_SAFE_CALL(hipFree(rest_vertexes));
    CUDA_SAFE_CALL(hipFree(xTilta));
    CUDA_SAFE_CALL(hipFree(fb));
    CUDA_SAFE_CALL(hipFree(shape_grads));
    CUDA_SAFE_CALL(hipFree(tetrahedras));
    CUDA_SAFE_CALL(hipFree(tempTetrahedras));
    CUDA_SAFE_CALL(hipFree(volum));
    CUDA_SAFE_CALL(hipFree(masses));
    CUDA_SAFE_CALL(hipFree(lengthRate));
    CUDA_SAFE_CALL(hipFree(volumeRate));
    CUDA_SAFE_CALL(hipFree(DmInverses));
    CUDA_SAFE_CALL(hipFree(Constraints));
    CUDA_SAFE_CALL(hipFree(tempMat3x3));
    CUDA_SAFE_CALL(hipFree(MChash));
    CUDA_SAFE_CALL(hipFree(tempDouble));
    CUDA_SAFE_CALL(hipFree(BoundaryType));
    CUDA_SAFE_CALL(hipFree(tempBoundaryType));

    CUDA_SAFE_CALL(hipFree(totalForce));
    CUDA_SAFE_CALL(hipFree(targetIndex));
    CUDA_SAFE_CALL(hipFree(targetVert));
    CUDA_SAFE_CALL(hipFree(triDmInverses));
    CUDA_SAFE_CALL(hipFree(area));
    CUDA_SAFE_CALL(hipFree(triangles));

    CUDA_SAFE_CALL(hipFree(tri_edges));
    CUDA_SAFE_CALL(hipFree(tri_edge_adj_vertex));

    CUDA_SAFE_CALL(hipFree(body_id_to_boundary_type));
    CUDA_SAFE_CALL(hipFree(point_id_to_body_id));
    CUDA_SAFE_CALL(hipFree(tet_id_to_body_id));

    CUDA_SAFE_CALL(hipFree(svd3x2F));
    CUDA_SAFE_CALL(hipFree(svd3x2U));
    CUDA_SAFE_CALL(hipFree(svd3x2V));
    CUDA_SAFE_CALL(hipFree(svd3x2S));
}
